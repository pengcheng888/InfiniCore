#include "hip/hip_runtime.h"
#include "cub_algorithms.cuh"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>

namespace infini_cub {

hipError_t cub_DeviceReduce_ArgMax(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *logits,
    hipcub::KeyValuePair<int, __hip_bfloat16> *kv_pair,
    int n,
    hipStream_t stream) {

    return hipcub::DeviceReduce::ArgMax(workspace_ptr, workspace_len, logits, kv_pair, n, stream);
}

hipError_t cub_DeviceReduce_ArgMax(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *logits,
    hipcub::KeyValuePair<int, half> *kv_pair,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceReduce::ArgMax(workspace_ptr, workspace_len, logits, kv_pair, n, stream);
}

hipError_t cub_DeviceReduce_ArgMax(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *logits,
    hipcub::KeyValuePair<int, float> *kv_pair,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceReduce::ArgMax(workspace_ptr, workspace_len, logits, kv_pair, n, stream);
}

hipError_t cub_DeviceReduce_ArgMax(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *logits,
    hipcub::KeyValuePair<int, double> *kv_pair,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceReduce::ArgMax(workspace_ptr, workspace_len, logits, kv_pair, n, stream);
}
} // namespace infini_cub

namespace infini_cub {

hipError_t cub_DeviceScan_InclusiveSum(
    void *workspace_ptr,
    size_t &workspace_len,
    __hip_bfloat16 *data,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceScan::InclusiveSum(workspace_ptr, workspace_len, data, data, n, stream);
}

hipError_t cub_DeviceScan_InclusiveSum(
    void *workspace_ptr,
    size_t &workspace_len,
    half *data,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceScan::InclusiveSum(workspace_ptr, workspace_len, data, data, n, stream);
}

hipError_t cub_DeviceScan_InclusiveSum(
    void *workspace_ptr,
    size_t &workspace_len,
    float *data,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceScan::InclusiveSum(workspace_ptr, workspace_len, data, data, n, stream);
}

hipError_t cub_DeviceScan_InclusiveSum(
    void *workspace_ptr,
    size_t &workspace_len,
    double *data,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceScan::InclusiveSum(workspace_ptr, workspace_len, data, data, n, stream);
}
} // namespace infini_cub

namespace infini_cub {
// --------------------------------------------------------------
// --------------------------------------------------------------
// --------------------------------------------------------------
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const uchar *val_in,
    uchar *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const int8_t *val_in,
    int8_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const ushort *val_in,
    ushort *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const short *val_in,
    short *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const uint *val_in,
    uint *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const int *val_in,
    int *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const ulong *val_in,
    ulong *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const long *val_in,
    long *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

#ifdef _WIN64
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const __hip_bfloat16 *key_in,
    __hip_bfloat16 *key_out,
    const ptrdiff_t *val_in,
    ptrdiff_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}
#endif

// --------------------------------------------------------------
// --------------------------------------------------------------
// --------------------------------------------------------------
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const uchar *val_in,
    uchar *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const int8_t *val_in,
    int8_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const ushort *val_in,
    ushort *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const short *val_in,
    short *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const uint *val_in,
    uint *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const int *val_in,
    int *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const ulong *val_in,
    ulong *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const long *val_in,
    long *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}

#ifdef _WIN64
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const half *key_in,
    half *key_out,
    const ptrdiff_t *val_in,
    ptrdiff_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(half) * 8,
        stream);
}
#endif

// --------------------------------------------------------------
// --------------------------------------------------------------
// --------------------------------------------------------------
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const uchar *val_in,
    uchar *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const int8_t *val_in,
    int8_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const ushort *val_in,
    ushort *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const short *val_in,
    short *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const uint *val_in,
    uint *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const int *val_in,
    int *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const ulong *val_in,
    ulong *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const long *val_in,
    long *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}

#ifdef _WIN64
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const float *key_in,
    float *key_out,
    const ptrdiff_t *val_in,
    ptrdiff_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(float) * 8,
        stream);
}
#endif

// --------------------------------------------------------------
// --------------------------------------------------------------
// --------------------------------------------------------------
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const uchar *val_in,
    uchar *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const int8_t *val_in,
    int8_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const ushort *val_in,
    ushort *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const short *val_in,
    short *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const uint *val_in,
    uint *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const int *val_in,
    int *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const ulong *val_in,
    ulong *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(
        workspace_ptr, workspace_len,
        key_in, key_out,
        val_in, val_out,
        n,
        0, sizeof(double) * 8,
        stream);
}

hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const long *val_in,
    long *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(workspace_ptr, workspace_len,
                                                     key_in, key_out,
                                                     val_in, val_out,
                                                     n,
                                                     0, sizeof(double) * 8,
                                                     stream);
}

#ifdef _WIN64
hipError_t cub_DeviceRadixSort_SortPairsDescending(
    void *workspace_ptr,
    size_t &workspace_len,
    const double *key_in,
    double *key_out,
    const ptrdiff_t *val_in,
    ptrdiff_t *val_out,
    int n,
    hipStream_t stream) {
    return hipcub::DeviceRadixSort::SortPairsDescending(workspace_ptr, workspace_len,
                                                     key_in, key_out,
                                                     val_in, val_out,
                                                     n,
                                                     0, sizeof(double) * 8,
                                                     stream);
}
#endif

} // namespace infini_cub
